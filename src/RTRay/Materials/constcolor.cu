#include "hip/hip_runtime.h"
#include "../rttypes.h"

#include <optix_math.h>

rtDeclareVariable(float3, color, , );
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

RT_PROGRAM void closest_hit_radiance()
{
  prd_radiance.result = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal))*0.5f + 0.5f;;
}