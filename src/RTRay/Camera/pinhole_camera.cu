#include "hip/hip_runtime.h"
/* Sergeev Artemiy
 * Realtime ray tracing system for rendering photo realistic scenes
 * Ray generation module
 * SPbSTU, IAMM, 2014
 */

#include <optix.h>
#include <optix_math.h>
#include <optixu/optixu_math_namespace.h>

#include "../rttypes.h"

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

rtBuffer<float4, 2> output_buffer;

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );

rtDeclareVariable(float3, cam_dir, , );
rtDeclareVariable(float3, cam_right, , );
rtDeclareVariable(float3, cam_up, , );
rtDeclareVariable(float3, cam_pos, , );

rtDeclareVariable(float, screen_half_width, , );
rtDeclareVariable(float, screen_half_height, , );

rtDeclareVariable(rtObject, top_object, , );

RT_PROGRAM void pinhole_camera()
{
  float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;
  float3 ray_origin = cam_pos;
  float3 ray_direction = normalize(d.x * cam_right + d.y * cam_up + cam_dir);
  optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;

  rtTrace(top_object, ray, prd);

  output_buffer[launch_index] = make_float4(prd.result, 1);
}

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(float3, bg_color, , );
RT_PROGRAM void miss()
{
  prd_radiance.result = bg_color;
}

rtDeclareVariable(float3, bad_color, , );
RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  output_buffer[launch_index] = make_float4(bad_color, 1);
  rtPrintf("Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
}
